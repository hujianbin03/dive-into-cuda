#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "../include/utils.h"

void sumMatrixOnHost(float *A, float *B, float *C, const int nx, const int ny){
    float *ia = A;
    float *ib = B;
    float *ic = C;

    for (int iy=0; iy<ny; iy++){
        for(int ix=0; ix<nx; ix++){
            ic[ix] = ia[ix] + ib[ix];
        }
        ia += nx;
        ib += nx;
        ic += nx;
    }
}

__global__ void sumMatrixOnGPU2D(float *MatA, float *MatB, float *MatC, int nx, int ny){
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;

    if (ix < nx && iy < ny){
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}

int main(int argc, char **argv){
    printf("%s 开始...\n", argv[0]);

    // 设置设备
    initDevice(0);

    // 设置矩阵维度
    int nx = 1 <<14;
    int ny = 1 <<14;
    int nxy = nx * ny;
    int nBytes = nxy * sizeof(float);
    printf("矩阵大小为: nx %d ny %d\n", nx, ny);

    // 主机申请内存
    float *h_A, *h_B, *hostRef, *gpuRef;
    h_A = (float *)malloc(nBytes);
    h_B = (float *)malloc(nBytes);
    hostRef = (float *)malloc(nBytes);
    gpuRef = (float *)malloc(nBytes);

    // 初始化矩阵
    double iStart = cpuSecond();
    initialData(h_A, nxy);
    initialData(h_B, nxy);
    double iElaps = cpuSecond() - iStart;

    // 初始化内存
    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    // 主机计算矩阵相加
    iStart = cpuSecond();
    sumMatrixOnHost(h_A, h_B, hostRef, nx, ny);
    iElaps = cpuSecond() - iStart;

    // 设备申请内存
    float *d_MatA, *d_MatB, *d_MatC;
    hipMalloc((void **)&d_MatA, nBytes);
    hipMalloc((void **)&d_MatB, nBytes);
    hipMalloc((void **)&d_MatC, nBytes);

    // 将数据从主机端传输到设备端
    hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_MatB, h_B, nBytes, hipMemcpyHostToDevice);

    // 设置线程块，线程格
    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((nx+block.x-1)/block.x, (ny+block.y-1)/block.y);

    // 执行核函数
    iStart = cpuSecond();
    sumMatrixOnGPU2D <<<grid, block>>>(d_MatA, d_MatB, d_MatC, nx, ny);
    hipDeviceSynchronize();
    iElaps = cpuSecond() - iStart;
    printf("GPU Execution configuration<<<(%d,%d),(%d,%d)>>> Time elapsed %f sec\n",
        grid.x,grid.y,block.x,block.y,iElaps); 

    // 将设备端结果传到主机端
    hipMemcpy(gpuRef, d_MatC, nBytes, hipMemcpyDeviceToHost);

    // 对比主机和设备结果
    checkResult(hostRef, gpuRef, nxy);

    // 释放内存
    hipFree(d_MatA);
    hipFree(d_MatB);
    hipFree(d_MatC);
    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);

    hipDeviceReset();
    return 0;
} 