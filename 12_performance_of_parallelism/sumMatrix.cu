#include "hip/hip_runtime.h"
#include "../include/utils.h"
#include <hip/hip_runtime.h>


void sumMatrixOnHost(float *A, float *B, float *C, const int nx, const int ny)
{
    float *ia = A;
    float *ib = B;
    float *ic = C;

    for(int iy = 0; iy < ny; iy++)
    {
        for(int ix = 0; ix < nx; ix++)
        {
            ic[ix] = ia[ix] + ib[ix];
        }   
        ia += nx;
        ib += nx;
        ic += nx;
    }
    return;
}

__global__ void sumMatrixOnGPU2D(float *A, float *B, float *C, int NX, int NY)
{
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int idx = iy * NX + ix;

    if (ix < NX && iy < NY)
    {
        C[idx] = A[idx] + B[idx];
    }
}

int main(int argc, char **argv)
{
    // 初始化设备
    initDevice(0);

    // 设置矩阵大小
    int nx = 1 << 14;
    int ny = 1 << 14;

    int nxy = nx * ny;
    int nBytes = nxy * sizeof(float);

    // 申请主机内存
    float *h_A, *h_B, *hostRef, *gpuRef;
    h_A = (float *)malloc(nBytes);
    h_B = (float *)malloc(nBytes);
    hostRef = (float *)malloc(nBytes);
    gpuRef = (float *)malloc(nBytes);

    // 在主机端初始化数据
    double iStart = seconds();
    initialData(h_A, nxy);
    initialData(h_B, nxy);
    double iElaps = seconds() - iStart;

    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    // 在主机端计算结果
    iStart = seconds();
    sumMatrixOnHost (h_A, h_B, hostRef, nx, ny);
    iElaps = seconds() - iStart;

    // 申请设备端内存
    float *d_MatA, *d_MatB, *d_MatC;
    CHECK(hipMalloc((void **)&d_MatA, nBytes));
    CHECK(hipMalloc((void **)&d_MatB, nBytes));
    CHECK(hipMalloc((void **)&d_MatC, nBytes));

    // 将数据从主机端发送到设备端
    CHECK(hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_MatB, h_B, nBytes, hipMemcpyHostToDevice));

    // 设置内核配置
    int dimx = 32;
    int dimy = 32;

    // 可以通过命令行设置内核
    if (argc > 2)
    {
        dimx = atoi(argv[1]);
        dimy = atoi(argv[2]);
    }

    dim3 block(dimx, dimy);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

    // 执行内核函数
    CHECK(hipDeviceSynchronize());
    iStart = seconds();
    sumMatrixOnGPU2D<<<grid, block>>>(d_MatA, d_MatB, d_MatC, nx, ny);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;
    printf("sumMatrixOnGPU2D <<<(%d,%d), (%d,%d)>>> elapsed %lf ms\n", grid.x,
           grid.y,
           block.x, block.y, iElaps);
    CHECK(hipGetLastError());

    // 复制设备端代码到主机端
    CHECK(hipMemcpy(gpuRef, d_MatC, nBytes, hipMemcpyDeviceToHost));

    // 检查设备端和主机端结果
    checkResult(hostRef, gpuRef, nxy);

    // 释放设备端内存
    CHECK(hipFree(d_MatA));
    CHECK(hipFree(d_MatB));
    CHECK(hipFree(d_MatC));

    // 释放主机端内存
    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);

    // 复位设备端
    CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}